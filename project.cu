#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
extern "C" {
	#include "header.h"
	#include "utilities.h"
}

//Aggregation function



//Combination function
__global__ void combination(float* in_feature, int fea_row, int fea_col,  float* weight, float* bias, int para_in, int para_out, float* out_feature, bool relu ){
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    //For output feature matrix
    int x = tx + blockDim.x * bx; //Nodes
    int y = ty + blockDim.y * by; //Out Features
    float val=0;

    if(x<fea_col && y<para_out){
	    out_feature[y*fea_col+x] = bias[y];
	    for (int k = 0; k < para_in; ++k){
		    val += in_feature[k*fea_col + x] * weight[k*para_out + y];
		 //   printf("%d %lf\n",k,in_feature[k*fea_col + x]); //, weight[k*para_out + y]);
	    }
	   out_feature[y*fea_col+x] += val;
	   if(relu) out_feature[y*fea_col+x] = MAX(0.00000, out_feature[y*fea_col+x]);
    }
    //__syncthreads();
}

//Analyzer function


//Testing function
__global__ void check(){
   // printf("Thread %d %d from block %d %d \n",threadIdx.x,threadIdx.y,blockIdx.x, blockIdx.y);
}

int main(int argc, char const *argv[]) {
	if ((argc != 2) || ((strcmp(argv[1], "cora") != 0) && (strcmp(argv[1], "citeseer") != 0) && (strcmp(argv[1], "reddit") != 0))) {
		printf("ERROR: usage \"%s [cora|citeseer|reddit]\"\n", argv[0]);
		return -1;
	}
	GCN_t GCN_c = GCN_parser((char*)argv[1]);
	feature_t feature_c;	
	//CUDA Code section
	//Add commands for profiling -> Look into this -> Lec 17.pdf
	//Aggregation kernal

	//Combination kernal
	//This is only for testing 
	feature_c = aggregation(GCN_c.graph_c, GCN_c.feature_c);

	//Timing 
	struct timeval stop, start;
	gettimeofday(&start, NULL);
	
	//Get parameter sizes
	int l1_para_in = GCN_c.l1_parameter_c.in_feature_num;
	int l1_para_out =GCN_c.l1_parameter_c.out_feature_num;

	//Define and allocate  outputs for the combination kernal
	float *device_parameter_weight, *device_parameter_bias;
	hipMalloc((void**)&device_parameter_weight, (l1_para_in*l1_para_out)*sizeof(float));
	hipMalloc((void**)&device_parameter_bias, (l1_para_out)*sizeof(float));
	hipMemcpy(device_parameter_weight,GCN_c.l1_parameter_c.weights[0] , (l1_para_in*l1_para_out)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_parameter_bias, GCN_c.l1_parameter_c.biasses, (l1_para_out)*sizeof(float), hipMemcpyHostToDevice);

	//This doesn't assume that the output of the aggregation kernal is stored in CUDA	
	//Make sure that this allocation is not needed
	float *in_feature;
	hipMalloc((void**)&in_feature, (feature_c.feature_num*feature_c.node_num)*sizeof(float));
	hipMemcpy(in_feature,feature_c.features[0] , (feature_c.feature_num*feature_c.node_num)*sizeof(float), hipMemcpyHostToDevice);
		
	//Define and allocate the output of the combination kernal
	float *out_feature;
	hipMalloc((void**)&out_feature, (feature_c.node_num*l1_para_out)*sizeof(float)); 

	//Define the grid and block sizes and launch the kernal
	dim3 Dg( ceil(GCN_c.spec_c.nodes/32.0),ceil(l1_para_out/32.0)) ;
        dim3 Db(32,32,1);
	combination<<<Dg,Db>>>(in_feature,feature_c.feature_num,feature_c.node_num, device_parameter_weight,device_parameter_bias, l1_para_in, l1_para_out,out_feature,true);
	hipDeviceSynchronize();
        
        gettimeofday(&stop, NULL);
	float secs1 = (double)(stop.tv_usec - start.tv_usec) / 1000000 + (double)(stop.tv_sec - start.tv_sec);
	
	//Checking the solution of the combination kernal
	//Remove later
	gettimeofday(&start, NULL);
	feature_t  feature_check = combination(feature_c, GCN_c.l1_parameter_c, true);
	gettimeofday(&stop, NULL);
	float secs2 = (double)(stop.tv_usec - start.tv_usec) / 1000000 + (double)(stop.tv_sec - start.tv_sec);
	
       	gettimeofday(&start, NULL);
	//Update feature_c and copy the feaatures values back into CPU
        feature_c.feature_num = l1_para_out;
	hipMemcpy(feature_c.features[0],out_feature,(feature_c.node_num*l1_para_out)*sizeof(float), hipMemcpyDeviceToHost); 
	gettimeofday(&stop, NULL);
	float secs3 = (double)(stop.tv_usec - start.tv_usec) / 1000000 + (double)(stop.tv_sec - start.tv_sec);

	//Testing the combination kernal 
	// Remove later	
        //for(int i=0; i<100; i++){
        //      printf("%d %lf %lf \n",i,feature_check.features[4][i], feature_c.features[4][i]); //feature_c.features[13][i]);
        //}

	printf("CPU Time: %f sec\n",secs2);
	printf("GPU Time: %f sec\n",secs1+secs3);

	hipFree(device_parameter_weight);
	hipFree(device_parameter_bias);
	hipFree(in_feature);
	hipFree(out_feature);
	//Aggregation kernal


        //Combination kernal
	
	
	//Analyzer kernal
	feature_c = aggregation(GCN_c.graph_c, feature_c);
        
	//Timing
        gettimeofday(&start, NULL);

        //Get parameter sizes
        int l2_para_in = GCN_c.l2_parameter_c.in_feature_num;
        int l2_para_out =GCN_c.l2_parameter_c.out_feature_num;

        //Define and allocate  outputs for the combination kernal
        hipMalloc((void**)&device_parameter_weight, (l2_para_in*l2_para_out)*sizeof(float));
        hipMalloc((void**)&device_parameter_bias, (l2_para_out)*sizeof(float));
        hipMemcpy(device_parameter_weight,GCN_c.l2_parameter_c.weights[0] , (l2_para_in*l2_para_out)*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_parameter_bias, GCN_c.l2_parameter_c.biasses, (l2_para_out)*sizeof(float), hipMemcpyHostToDevice);

        //This doesn't assume that the output of the aggregation kernal is stored in CUDA
        //Make sure that this allocation is not needed
        hipMalloc((void**)&in_feature, (feature_c.feature_num*feature_c.node_num)*sizeof(float));
        hipMemcpy(in_feature,feature_c.features[0] , (feature_c.feature_num*feature_c.node_num)*sizeof(float), hipMemcpyHostToDevice);

        //Define and allocate the output of the combination kernal
        hipMalloc((void**)&out_feature, (feature_c.node_num*l2_para_out)*sizeof(float));

        //Define the grid and block sizes and launch the kernal
        dim3 Dg2( ceil(GCN_c.spec_c.nodes/32.0),ceil(l2_para_out/32.0)) ;
        combination<<<Dg2,Db>>>(in_feature,feature_c.feature_num,feature_c.node_num, device_parameter_weight,device_parameter_bias, l2_para_in, l2_para_out,out_feature,false);
        hipDeviceSynchronize();

        gettimeofday(&stop, NULL);
        secs1 = (double)(stop.tv_usec - start.tv_usec) / 1000000 + (double)(stop.tv_sec - start.tv_sec);

        //Checking the solution of the combination kernal
        //Remove later
        gettimeofday(&start, NULL);
        feature_check = combination(feature_c, GCN_c.l2_parameter_c, false);
        gettimeofday(&stop, NULL);
        secs2 = (double)(stop.tv_usec - start.tv_usec) / 1000000 + (double)(stop.tv_sec - start.tv_sec);

        gettimeofday(&start, NULL);
        //Update feature_c and copy the feaatures values back into CPU
        feature_c.feature_num = l2_para_out;
        hipMemcpy(feature_c.features[0],out_feature,(feature_c.node_num*l2_para_out)*sizeof(float), hipMemcpyDeviceToHost);
        gettimeofday(&stop, NULL);
        secs3 = (double)(stop.tv_usec - start.tv_usec) / 1000000 + (double)(stop.tv_sec - start.tv_sec);

        //Testing the combination kernal
        // Remove later
        //for(int i=0; i<100; i++){
        //      printf("%d %lf %lf \n",i,feature_check.features[4][i], feature_c.features[4][i]); //feature_c.features[13][i]);
        //}

        printf("CPU Time: %f sec\n",secs2);
        printf("GPU Time: %f sec\n",secs1+secs3);

        hipFree(device_parameter_weight);
        hipFree(device_parameter_bias);
        hipFree(in_feature);
        hipFree(out_feature);

	
	//feature_c = combination(feature_c, GCN_c.l2_parameter_c, false);
        analyzer(feature_c, GCN_c.label_c);
	return 0;
}
